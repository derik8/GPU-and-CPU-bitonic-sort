#include "hip/hip_runtime.h"
#include<iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> 
#include<string.h>
#include<string>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include<vector>
#include <algorithm>  


#include <fstream>
#include<ctime>
#include<chrono>



__global__  static void bitonicSequence(int * values, int k, int j);

void sort_GPU(int * mass,int size);
void sort_CPU(int * mass, int size);

void write(std::ofstream &file, int * mass, int size, std::string s);
void generate_mass(  int * mass, int size);
void read_mass(std::ifstream &file, int * mass,  int size);
void write_nbs(std::ofstream & file,int size);

int main(int argc, char* argv[])
{
	srand(time(0));
	
	std::vector<std::string> vec_arg;

	
	for (int i = 0; i < argc; i++) {
		
		vec_arg.push_back(std::string(argv[i]));
	}

	std::vector<std::string>::iterator d = find(vec_arg.begin(), vec_arg.end(), "-d");
	std::vector<std::string>::iterator s = find(vec_arg.begin(), vec_arg.end(), "-s");
	std::vector<std::string>::iterator i = find(vec_arg.begin(), vec_arg.end(), "-i");
	std::vector<std::string>::iterator o = find(vec_arg.begin(), vec_arg.end(), "-o");
	std::vector<std::string>::iterator n = find(vec_arg.begin(), vec_arg.end(), "-n");
	
	
//size mass
	int size_mass = stoi(*(++s));

//mass
	int *mass = new int[size_mass];
	

//read or generate data
	if(i!=vec_arg.end()){
		std::ifstream file_read(*(++i));
		read_mass(file_read, mass, size_mass);
	}
	else {
		generate_mass(mass, size_mass);
	}
	
	

	std::cout << "calculation" << std::endl;

// start sort GPU or CPU
	if(*(++d)=="GPU")
		sort_GPU(mass, size_mass);
	else
		sort_CPU(mass, size_mass);
		
	
	if (o != vec_arg.end())	{
		std::cout << "write to file..."<< std::endl;
		std::ofstream file_write(*(++o));
		write(file_write, mass, size_mass, *d);
		
	}
	
	if (n != vec_arg.end()){
		std::cout << "write to file..."<< std::endl;
		std::ofstream file_write(*(++n));
		write_nbs(file_write, size_mass);	
		
	}
	

	

	delete[] mass;
	system("pause");
	return 0;
}


__global__  static void bitonicSequence(int * values, int k, int j)
{
	int temp;
	int tid = blockIdx.x * blockDim.x + threadIdx.x;


	unsigned int ixj = tid ^ j;
	if (ixj > tid) {
		if ((tid & k) == 0)
		{
			if (values[tid] > values[ixj]) {
				temp = values[tid];
				values[tid] = values[ixj];
				values[ixj] = temp;
			}
		}
		else
		{
			if (values[tid] < values[ixj]) {
				temp = values[tid];
				values[tid] = values[ixj];
				values[ixj] = temp;
			}
		}

	}
}

void sort_GPU(int * mass, int size)
{
	


	std::cout << "GPU" << std::endl;

	
	int * dvalues;
	hipMalloc(&dvalues, sizeof(int) * size);
	hipMemcpy(dvalues, mass, sizeof(int) * size, hipMemcpyHostToDevice);

	int threads = 128;
	int blocks = ceil(size / threads);

	if (size < threads)
	{
		threads = size;
		blocks = 1;
	}
	
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	


	for (unsigned int k = 2; k <= size; k *= 2) {
		for (unsigned int j = k / 2; j > 0; j /= 2) {
			bitonicSequence << <blocks, threads >> > (dvalues, k, j);
			
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	std::cout << "time: " << gpuTime <<" milliseconds"<<  std::endl;
	

	hipMemcpy(mass, dvalues, sizeof(int) * size, hipMemcpyDeviceToHost);


	hipFree(dvalues);

}

void sort_CPU(int * mass, int size)
{
	
	std::cout << "CPU" << std::endl;

	auto start = std::chrono::high_resolution_clock::now();
	for (unsigned int k = 2; k <= size; k *= 2) 
	{
		for (unsigned int j = k / 2; j > 0; j /= 2)
		{
			for (int i = 0; i < size; i++)
			{
				unsigned int ixj = i ^ j;
				if (ixj > i) {
					
					
					if ((i & k) == 0)
					{
						if (mass[i] > mass[ixj])
							std::swap(mass[i], mass[ixj]);			

					}
					else
					{
						if (mass[i] < mass[ixj])
							std::swap(mass[i], mass[ixj]);
						
					}					
				}
			}			
		}
	}
	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	long long microseconds = std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
	std::cout << "time: " << (double)microseconds/1000 << " milliseconds" << std::endl;
}

void write(std::ofstream &file, int * mass, int size,std::string s)
{
	if (file.is_open())
	{
		file << s << std::endl;
		for (int i = 0; i < size; i++)
		{
			file << mass[i] << std::endl;
		}
		file.close();
	}
	else std::cout << "Unable to open file";
}

void generate_mass(int * mass, int size)
{
	for (int i = 0; i < size; i++)
	{
		mass[i] = rand();
	}
	
}

void read_mass(std::ifstream &file,int *mass,  int size)
{
	
	if (file.is_open())
	{
		for (int i = 0; i < size; i++)
		{
			file >> mass[i];
		}
		
		file.close();
	}

	else std::cout << "Unable to open file";
}

void write_nbs(std::ofstream & file, int size)
{
	int *mass_index_BN = new int[size];

	for (unsigned int k = 2; k <= size; k *= 2)	{
		for (unsigned int j = k / 2; j > 0; j /= 2)	{
			for (int i = 0; i < size; i++){

				unsigned int ixj = i ^ j;

				if (ixj > i) {
					mass_index_BN[i] = ixj + 1;
					mass_index_BN[ixj] = i + 1;
					if ((i & k) == 0)
						mass_index_BN[ixj] *= -1;
					else
						mass_index_BN[i] *= -1;
				}
				
			}
			for (size_t i = 0; i < size; i++){
				file << mass_index_BN[i] << " ";
			}
			file << std::endl;
		}
		file << std::endl;
	}

	file.close();
	delete[]mass_index_BN;	
}
